#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    const int n = 1000000; // Size of the vectors
    float *x, *y, *d_x, *d_y;
    float alpha = 1.0;

    // Allocate host memory
    x = (float*)malloc(n * sizeof(float));
    y = (float*)malloc(n * sizeof(float));

    // Initialize host arrays
    for (int i = 0; i < n; ++i) {
        x[i] = static_cast<float>(i);
        y[i] = static_cast<float>(i);
    }

    // Allocate device memory
    hipMalloc(&d_x, n * sizeof(float)); 
    hipMalloc(&d_y, n * sizeof(float));

    // Copy host memory to device
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);

    // Create a handle for CUBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Perform the vector addition: y = alpha * x + y
    hipblasSaxpy(handle, n, &alpha, d_x, 1, d_y, 1);

    // Copy result back to host
    hipMemcpy(y, d_y, n * sizeof(float), hipMemcpyDeviceToHost);
    
    int N = 1; 
    std::cout << "\n---------------------------\n";
    std::cout << "__SUCCESS__\n";
    std::cout << "---------------------------\n";
    std::cout << "N                 = %d\n" <<  N;
    //printf("Threads Per Block = %d\n", thr_per_blk);
    //printf("Blocks In Grid    = %d\n", blk_in_grid);
    std::cout << "---------------------------\n\n";


    // Clean up resources
    hipFree(d_x);
    hipFree(d_y);
    hipblasDestroy(handle);
    free(x);
    free(y);

    return 0;
}